
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <ctime>
#include <cmath>
#define N 5

__global__ void RMSNorm(float*x,float* out){
    int row_start = threadIdx.x * N;
    int end = row_start + N; 
    float eps = 1e-6;
    float rms_x = 0.0;
    for(int i=row_start;i<end;i++){
        rms_x += x[i]*x[i];
    }
    rms_x = sqrt(rms_x / N);
    for(int i=row_start;i<end;i++){
        out[i] = x[i] / (rms_x + eps);
    }

}

int main(){
    std::default_random_engine e;
    std::normal_distribution<float> u(0,1); // 均值为0，标准差为1
    e.seed(time(0));

    float X[N][N],O[N][N];
    float *x,*out;
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            X[i][j] = u(e);
            printf("%f ",X[i][j]);
        }
        printf("\n");
    }
    int bytes_size = sizeof(float)*N*N;
    hipMalloc(&x,bytes_size);
    hipMalloc(&out,bytes_size);

    hipMemcpy(x,X,bytes_size,hipMemcpyHostToDevice);

    RMSNorm<<<1, N>>>(x,out);
    hipMemcpy(O,out,bytes_size,hipMemcpyDeviceToHost);
    printf("RMSNorm....\n");
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            printf("%f ",O[i][j]);
        }
        printf("\n");
    }
    hipFree(out);


    return 0;
}