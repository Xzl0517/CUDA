
#include <hip/hip_runtime.h>
#include <iostream>
#define  N 20
//1 个block 1个thread
__global__ void vector_add_1(int *a,int *b,int *c,int n){
    int id = 0;
    while(id<n){
        c[id] = a[id] + b[id];
        id+=1;
    }
}
// 1个block 多个thread
__global__ void vector_add_2(int *a,int *b,int *c,int n){
    int tid = threadIdx.x;
    int offs = blockDim.x;
    while(tid<n){
        c[tid] = a[tid] + b[tid];
        tid+=offs;
    }
}
// 多个block 多个thread
__global__ void vector_add_3(int *a,int *b,int *c,int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int offs = gridDim.x * blockDim.x;
    while(tid<n){
        c[tid] = a[tid] + b[tid];
        tid += offs;
    }
}
int main(){
    int A[N],B[N],C[N];
    int *a_cuda,*b_cuda,*c_cuda;

    for(int i=0;i<N;i++){
        A[i]=i;
        B[i]=i;
    }

    hipMalloc(&a_cuda,sizeof(int) * N);
    hipMemcpy(a_cuda, A, sizeof(int) * N , hipMemcpyHostToDevice);
    
    hipMalloc(&b_cuda,sizeof(int) * N);
    hipMemcpy(b_cuda, B, sizeof(int) * N , hipMemcpyHostToDevice);

    hipMalloc(&c_cuda,sizeof(int) * N);
    hipMemcpy(c_cuda, C, sizeof(int) * N , hipMemcpyHostToDevice);

    vector_add_1<<<1,1>>>(a_cuda,b_cuda,c_cuda,N);
    hipMemcpy(C, c_cuda, sizeof(int) * N , hipMemcpyDeviceToHost);
    for (int i=0;i<N;i++){
        printf("%d ",C[i]);
    }
    printf("\n");
    vector_add_2<<<1,5>>>(a_cuda,b_cuda,c_cuda,N);
    hipMemcpy(C, c_cuda, sizeof(int) * N , hipMemcpyDeviceToHost);
    for (int i=0;i<N;i++){
        printf("%d ",C[i]);
    }
    printf("\n");
    vector_add_3<<<2,5>>>(a_cuda,b_cuda,c_cuda,N);
    hipMemcpy(C, c_cuda, sizeof(int) * N , hipMemcpyDeviceToHost);
    for (int i=0;i<N;i++){
        printf("%d ",C[i]);
    }
    printf("\n");
    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(c_cuda);
    return 0;
}