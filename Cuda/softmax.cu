
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <ctime>
#include <cmath>
#define N 5
__global__ void softmax(float *x , float *out,int col){
    int row_start = threadIdx.x * N;
    int end = row_start + col; 
    float sum = 0.0;
    for(int i=row_start;i<end;i++){
        sum += exp(x[i]);
    }
    for(int i=row_start;i<end;i++){
        out[i] = exp(x[i]) / sum;
    }
}


int main(){
    std::default_random_engine e;
    std::normal_distribution<float> u(0,1); // 均值为0，标准差为1
    e.seed(time(0));

    float X[N][N],O[N][N];
    float *x,*out;
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            X[i][j] = u(e);
            printf("%f ",X[i][j]);
        }
        printf("\n");
    }
    int bytes_size = sizeof(float)*N*N;
    hipMalloc(&x,bytes_size);
    hipMalloc(&out,bytes_size);
    hipMemcpy(x,X,bytes_size,hipMemcpyHostToDevice);

    softmax<<<1,N>>>(x,out,N);

    hipMemcpy(O,out,bytes_size,hipMemcpyDeviceToHost);
    printf("softmax....\n");
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            printf("%f ",O[i][j]);
        }
        printf("\n");
    }
    hipFree(out);
    return 0;
}