
#include <hip/hip_runtime.h>
#include <iostream>
#define N 10
__global__ void matmul(int* a,int* b,int* c,int stride_AK){
    int row_id = blockIdx.x ;
    int col_id = threadIdx.x;
    int sum = 0;
    for (int i=0;i<stride_AK;i++){
        sum += a[row_id * blockDim.x + i] * b[i * blockDim.x + col_id];     
    }
    c[row_id * blockDim.x + col_id] = sum;

}


int main(){
    int A[N][N],B[N][N],C[N][N];
    int *a_cuda,*b_cuda,*c_cuda;

    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            A[i][j] = i;
            B[i][j] = j;
        }
    }
    
    int bytes_size = sizeof(int)*N*N;
    hipMalloc(&a_cuda,bytes_size);
    hipMalloc(&b_cuda,bytes_size);
    hipMalloc(&c_cuda,bytes_size);

    hipMemcpy(a_cuda,A,bytes_size,hipMemcpyHostToDevice);
    hipMemcpy(b_cuda,B,bytes_size,hipMemcpyHostToDevice);


    matmul<<<N,N>>>(a_cuda,b_cuda,c_cuda, N);
    
    hipMemcpy(C,c_cuda,bytes_size,hipMemcpyDeviceToHost);

    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            printf("%d ",C[i][j]);
        }
        printf("\n");
    }

    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(c_cuda);
    return 0;
}