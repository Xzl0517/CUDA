
#include <hip/hip_runtime.h>
#include <iostream>
#define N 10
__global__ void matadd(int* a,int* b,int* c){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    c[id] = a[id] + b[id];
}


int main(){
    int A[N][N],B[N][N],C[N][N];
    int *a_cuda,*b_cuda,*c_cuda;

    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            A[i][j] = i;
            B[i][j] = j;
        }
    }
    
    int bytes_size = sizeof(int)*N*N;
    hipMalloc(&a_cuda,bytes_size);
    hipMalloc(&b_cuda,bytes_size);
    hipMalloc(&c_cuda,bytes_size);

    hipMemcpy(a_cuda,A,bytes_size,hipMemcpyHostToDevice);
    hipMemcpy(b_cuda,B,bytes_size,hipMemcpyHostToDevice);

    matadd<<<N,N>>>(a_cuda,b_cuda,c_cuda);
    
    hipMemcpy(C,c_cuda,bytes_size,hipMemcpyDeviceToHost);

    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            printf("%d ",C[i][j]);
        }
        printf("\n");
    }

    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(c_cuda);
    return 0;
}