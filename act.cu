
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <ctime>
#include <cmath>
#define N 5
__global__ void Leaky_Relu(float *x, float alpha){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("id:%d\n", id);
    printf("threadidx:%d\n", threadIdx.x);
    if(x[id] < 0){
        x[id] *= alpha;
    }
}

__global__ void Relu(float *x){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(x[id]<0) x[id] = 0;
}

__global__ void Sigmoid(float *x){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    x[id] = 1.0 / (1.0 + exp(x[id]));
}

__global__ void Tanh(float *x){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    x[id] = (exp(x[id]) - exp(-x[id])) / (exp(x[id]) + exp(-x[id]));
}

__global__ void Gelu(float *x){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    float tanh_x = (exp(x[id]) - exp(-x[id])) / (exp(x[id]) + exp(-x[id]));
    float x_p = exp(tanh_x);
    float x_n = exp(-tanh_x);
    x[id] = 0.5 * x[id] *(1.0 + (x_p - x_n)/(x_p + x_n));
}

__global__ void Silu(float *x){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    x[id] = x[id] / (1.0 + exp(x[id]));
}


int main(){
    std::default_random_engine e;
    std::normal_distribution<float> u(0,1); // 均值为0，标准差为1
    e.seed(time(0));
    float A[N][N],B[N][N];
    float *A_cuda;

    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            A[i][j]=u(e);
            printf("%f ",A[i][j]);
        }
        printf("\n");
    }
    hipMalloc((void **)&A_cuda,sizeof(float)*N*N);
    hipMemcpy(A_cuda,A,sizeof(float)*N*N,hipMemcpyHostToDevice);

    //Leaky_Relu<<<N,N>>>(A_cuda, 0.01);
    //Relu<<<N,N>>>(A_cuda);
    //Sigmoid<<<N,N>>>(A_cuda);
    //Tanh<<<N,N>>>(A_cuda);
    Gelu<<<N,N>>>(A_cuda);

    hipMemcpy(B,A_cuda,sizeof(float)*N*N,hipMemcpyDeviceToHost);

    printf("act......\n");
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            printf("%f ",B[i][j]);
        }
        printf("\n");
    }

    hipFree(A_cuda);
    return 0;
}

